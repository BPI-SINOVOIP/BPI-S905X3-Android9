// expected-no-diagnostics

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

struct S {
  __host__ static void operator delete(void*, size_t) {}
  __device__ static void operator delete(void*, size_t) {}
};

__host__ __device__ void test(S* s) {
  // This shouldn't be ambiguous -- we call the host overload in host mode and
  // the device overload in device mode.
  delete s;
}

__host__ void operator delete(void *ptr) {}
__device__ void operator delete(void *ptr) {}

__host__ __device__ void test_global_delete(int *ptr) {
  // Again, there should be no ambiguity between which operator delete we call.
  ::delete ptr;
}
