
#include <hip/hip_runtime.h>
// Check that types, widths, etc. match on the host and device sides of CUDA
// compilations.  Note that we filter out long double, as this is intentionally
// different on host and device.

// RUN: %clang --cuda-host-only -nocudainc -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/i386-host-defines
// RUN: %clang --cuda-device-only -nocudainc -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/i386-device-defines
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)' %T/i386-host-defines   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-host-defines-filtered
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF|WIDTH\)' %T/i386-device-defines | grep -v '__LDBL\|_LONG_DOUBLE' > %T/i386-device-defines-filtered
// RUN: diff %T/i386-host-defines-filtered %T/i386-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/x86_64-host-defines
// RUN: %clang --cuda-device-only -nocudainc -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/x86_64-device-defines
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/x86_64-host-defines   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-host-defines-filtered
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/x86_64-device-defines | grep -v '__LDBL\|_LONG_DOUBLE' > %T/x86_64-device-defines-filtered
// RUN: diff %T/x86_64-host-defines-filtered %T/x86_64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/powerpc64-host-defines
// RUN: %clang --cuda-device-only -nocudainc -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null > %T/powerpc64-device-defines
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/powerpc64-host-defines   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/powerpc64-host-defines-filtered
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/powerpc64-device-defines | grep -v '__LDBL\|_LONG_DOUBLE' > %T/powerpc64-device-defines-filtered
// RUN: diff %T/powerpc64-host-defines-filtered %T/powerpc64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target nvptx-nvidia-cuda -x cuda -E -dM -o - /dev/null > %T/nvptx-host-defines
// RUN: %clang --cuda-device-only -nocudainc -target nvptx-nvidia-cuda -x cuda -E -dM -o - /dev/null > %T/nvptx-device-defines
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/nvptx-host-defines   | grep -v '__LDBL\|_LONG_DOUBLE' > %T/nvptx-host-defines-filtered
// RUN: grep 'define __[^ ]*\(TYPE\|MAX\|SIZEOF\|WIDTH\)' %T/nvptx-device-defines | grep -v '__LDBL\|_LONG_DOUBLE' > %T/nvptx-device-defines-filtered
// RUN: diff %T/nvptx-host-defines-filtered %T/nvptx-device-defines-filtered
