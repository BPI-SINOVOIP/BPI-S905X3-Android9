
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang -v -### --cuda-gpu-arch=sm_20 --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang -v -### --cuda-gpu-arch=sm_20 --sysroot=%S/Inputs/CUDA_80 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang -v -### --cuda-gpu-arch=sm_60 --sysroot=%S/Inputs/CUDA_80 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK

// The installation at Inputs/CUDA is CUDA 7.0, which doesn't support sm_60.
// RUN: %clang -v -### --cuda-gpu-arch=sm_60 --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// This should only complain about sm_60, not sm_35.
// RUN: %clang -v -### --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_35 \
// RUN:    --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60 --check-prefix=OK_SM35

// We should get two errors here, one for sm_60 and one for sm_61.
// RUN: %clang -v -### --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_61 \
// RUN:    --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60 --check-prefix=ERR_SM61

// We should still get an error if we pass -nocudainc, because this compilation
// would invoke ptxas, and we do a version check on that, too.
// RUN: %clang -v -### --cuda-gpu-arch=sm_60 -nocudainc --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// If with -nocudainc and -E, we don't touch the CUDA install, so we
// shouldn't get an error.
// RUN: %clang -v -### -E --cuda-device-only --cuda-gpu-arch=sm_60 -nocudainc \
// RUN:    --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK

// --no-cuda-version-check should suppress all of these errors.
// RUN: %clang -v -### --cuda-gpu-arch=sm_60 --sysroot=%S/Inputs/CUDA 2>&1 \
// RUN:    --no-cuda-version-check %s | \
// RUN:    FileCheck %s --check-prefix=OK

// OK-NOT: error: GPU arch

// OK_SM35-NOT: error: GPU arch sm_35

// We should only get one error per architecture.
// ERR_SM60: error: GPU arch sm_60 {{.*}}
// ERR_SM60-NOT: error: GPU arch sm_60

// ERR_SM61: error: GPU arch sm_61 {{.*}}
// ERR_SM61-NOT: error: GPU arch sm_61
